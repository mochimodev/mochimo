#include "hip/hip_runtime.h"
/**
 * @private
 * @headerfile peach.cuh <peach.cuh>
 * @copyright Adequate Systems LLC, 2018-2022. All Rights Reserved.
 * <br />For license information, please refer to ../LICENSE.md
 * @note A note on variable naming in this file...
 * [2] suffix implies dual stream variables; for maximum GPU utilisation
 * h_ prefix implies host memory; requires hipHostMalloc()
 * c_ prefix implies (constant) device memory
 * g_ prefix implies (global) device memory
 * d_ prefix implies device memory
 * *h_ pointers require hipHostMalloc();
 * *d_/*g_ pointers require hipMalloc();
*/

/* include guard */
#ifndef MOCHIMO_PEACH_CU
#define MOCHIMO_PEACH_CU


#include "peach.cuh"

/* internal support */
#include "error.h"

/* external support */
#include "extint.h"
#include "extmath.h"

/* external support -- Nighthash */
#include "blake2b.h"
#include "md2.h"
#include "md5.h"
#include "sha1.h"
#include "sha256.cu"
#include "sha3.h"

/**
 * @private
 * Definitions for embedding strings.
*/
#define cuSTRING(x) #x
#define cuSTR(x) cuSTRING(x)

/**
 * @private
 * Peach specific cuda error checking definition.
 * Destroys/frees resources on error.
*/
#define cuCHK(_cmd, _dev, _exec) \
   do { \
      hipError_t _cerr = _cmd; \
      if (_cerr != hipSuccess) { \
         int _n; hipGetDevice(&_n); \
         const char *_err = hipGetErrorString(_cerr); \
         perr("CUDA#%d->%s: %s", _n, cuSTR(_cmd), _err); \
         peach_free_cuda_device(_dev, DEV_FAIL); \
         _exec; \
      } \
   } while(0)

/* sm_61 performs MUCH better with the __constant__ qualifier */
#if __CUDA_ARCH__ == 610
   #define cuCONSTn860 __constant__
#else
   #define cuCONSTn860
#endif

/**
 * @private
 * Peach CUDA context. Managed internally by cross referencing parameters
 * of DEVICE_CTX passed to functions.
*/
typedef struct {
   nvmlDevice_t nvml_device;           /**< nvml device for monitoring */
   hipStream_t stream[2];             /**< asynchronous streams */
   SHA256_CTX *h_ictx[2], *d_ictx[2];  /**< sha256 ictx lists */
   BTRAILER *h_bt[2];                  /**< BTRAILER (current) */
   word64 *h_solve[2], *d_solve[2];    /**< solve seeds */
   word64 *d_map;                      /**< Peach Map */
   word32 *d_phash;                    /**< previous hash */
   int nvml_enabled;                   /**< Flags NVML capable */
} PEACH_CUDA_CTX;

/* pointer to peach CUDA context/s */
static PEACH_CUDA_CTX *PeachCudaCTX;

/**
 * @private
 * 256-bit Blake2b (w/ key) computation optimized for the Peach algorithm.
 * Places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param keylen Length of optional @a key input, in bytes
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_peach_blake2b(const word64 *in, size_t inlen, int keylen,
   word64 *out)
{
   /* Blake2b compression constant */
   cuCONSTn860 static word8 c_sigma[12][16] = {
      { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
      { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
      { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
      { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
      { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
      { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
      { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
      { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
      { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
      { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
      { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
      { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
   };

   /* blake2b_init - outlen is always 256-bits in Peach */
   word64 v[16];
   word64 state[8];
   word64 final[16];
   word64 t[2] = { 128, 0 };

   /* FAST-FORWARD state to known keylen states */
   if (keylen == 64) {
      state[0] = WORD64_C(0x00B8AA23C261EF69);
      state[1] = WORD64_C(0xD38AE6ABCA237B9E);
      state[2] = WORD64_C(0x67FB881E5EE89069);
      state[3] = WORD64_C(0x3E5B8BD06B58D002);
      state[4] = WORD64_C(0x252D3F68395AAE91);
      state[5] = WORD64_C(0xD25465E23C6C1B27);
      state[6] = WORD64_C(0x852B4CC2E13303B5);
      state[7] = WORD64_C(0x3F38B9FF245BE7C1);
   } else {
      state[0] = WORD64_C(0x63320ACE264383EB);
      state[1] = WORD64_C(0x012AF5FD045A2737);
      state[2] = WORD64_C(0xF4F49C55E6BE39DF);
      state[3] = WORD64_C(0x791C5BC8AFFB11A7);
      state[4] = WORD64_C(0xC9BCACC002C0EA21);
      state[5] = WORD64_C(0x8295B8ABE2FDEDD6);
      state[6] = WORD64_C(0xB711490E5F9F41C8);
      state[7] = WORD64_C(0x3F8E4D1D9EBEAF1A);
   }

   /* blake2b_update */
   for(; inlen > 128; inlen -= 128, in = &in[16]) {
      t[0] += 128;
      blake2b_compress_init(v, state, t, 0);
      blake2b_compress_rounds(v, in, c_sigma);
      blake2b_compress_set(v, state);
   }

   /* blake2b_final - somewhat conveniently (and exclusive to Peach)...
    * the remaining datalen will always be 36... */
   final[0] = in[0];
   final[1] = in[1];
   final[2] = in[2];
   final[3] = in[3];
   final[4] = (word64) ((word32 *) in)[8];
   final[5] = 0;
   final[6] = 0;
   final[7] = 0;
   final[8] = 0;
   final[9] = 0;
   final[10] = 0;
   final[11] = 0;
   final[12] = 0;
   final[13] = 0;
   final[14] = 0;
   final[15] = 0;

   t[0] += 36;
   blake2b_compress_init(v, state, t, 1);
   blake2b_compress_rounds(v, final, c_sigma);

   /* blake2b_output */
   out[0] = state[0] ^ v[0] ^ v[8];
   out[1] = state[1] ^ v[1] ^ v[9];
   out[2] = state[2] ^ v[2] ^ v[10];
   out[3] = state[3] ^ v[3] ^ v[11];
}  /* end cu_peach_blake2b() */

/**
 * @private
 * 128-bit MD2 computation optimized for the Peach algorithm.
 * Places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_peach_md2(const word64 *in, size_t inlen, word64 *out)
{
   /* MD2 transformation constant */
   cuCONSTn860 static word8 s[256] = {
      41, 46, 67, 201, 162, 216, 124, 1, 61, 54, 84, 161, 236, 240, 6,
      19, 98, 167, 5, 243, 192, 199, 115, 140, 152, 147, 43, 217, 188,
      76, 130, 202, 30, 155, 87, 60, 253, 212, 224, 22, 103, 66, 111, 24,
      138, 23, 229, 18, 190, 78, 196, 214, 218, 158, 222, 73, 160, 251,
      245, 142, 187, 47, 238, 122, 169, 104, 121, 145, 21, 178, 7, 63,
      148, 194, 16, 137, 11, 34, 95, 33, 128, 127, 93, 154, 90, 144, 50,
      39, 53, 62, 204, 231, 191, 247, 151, 3, 255, 25, 48, 179, 72, 165,
      181, 209, 215, 94, 146, 42, 172, 86, 170, 198, 79, 184, 56, 210,
      150, 164, 125, 182, 118, 252, 107, 226, 156, 116, 4, 241, 69, 157,
      112, 89, 100, 113, 135, 32, 134, 91, 207, 101, 230, 45, 168, 2, 27,
      96, 37, 173, 174, 176, 185, 246, 28, 70, 97, 105, 52, 64, 126, 15,
      85, 71, 163, 35, 221, 81, 175, 58, 195, 92, 249, 206, 186, 197,
      234, 38, 44, 83, 13, 110, 133, 40, 132, 9, 211, 223, 205, 244, 65,
      129, 77, 82, 106, 220, 55, 200, 108, 193, 171, 250, 36, 225, 123,
      8, 12, 189, 177, 74, 120, 136, 149, 139, 227, 99, 232, 109, 233,
      203, 213, 254, 59, 0, 29, 57, 242, 239, 183, 14, 102, 88, 208, 228,
      166, 119, 114, 248, 235, 117, 75, 10, 49, 68, 80, 180, 143, 237,
      31, 26, 219, 153, 141, 51, 159, 17, 131, 20
   };

   /* md2_init */
   word64 state[6] = { 0 };
   word64 checksum[2] = { 0 };
   word64 pad64;
   word8 pad;

   /* prepare padding */
   pad = 16 - (inlen & 0xf);
   pad64 = pad | pad << 8;
   pad64 = pad64 | pad64 << 16;
   pad64 = pad64 | pad64 << 32;

   /* md2_update */
   for (; inlen >= 16; inlen -= 16, in = &in[2]) {
      md2_transform_init64(state, in);
      md2_transform_checksum(((word8 *) checksum), ((word8 *) in), s);
      md2_transform_state(((word8 *) state), s);
   }

   /* md2_final - only 4 bytes left, so 12 remaining bytes are pad */
   state[4] = (state[2] = *((word32 *) in) | (pad64 << 32)) ^ state[0];
   state[5] = (state[3] = pad64) ^ state[1];
   /* final transform part1 */
   md2_transform_checksum(((word8 *) checksum), ((word8 *) &state[2]), s);
   md2_transform_state(((word8 *) state), s);
   /* final transform part2 */
   md2_transform_init64(state, checksum);
   md2_transform_state(((word8 *) state), s);

   /* MD2 hash = 128 bits, zero fill remaining... */
   out[0] = state[0];
   out[1] = state[1];
   out[2] = 0;
   out[3] = 0;
}  /* end cu_peach_md2 */

/**
 * @private
 * 128-bit MD5 computation optimized for the Peach algorithm.
 * Places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_peach_md5(const word32 *in, size_t inlen, word64 *out)
{
   /* md5_init */
   word32 final[16];
   word32 state[4] = { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476 };

   /* prepare bitlen in final data */
   final[14] = inlen << 3;

   /* md5_update */
   for (; inlen >= 64; inlen -= 64, in = &in[16]) {
      md5_tranform_unrolled(state, in);
   }

   /* md5_final - somewhat conveniently (and exclusive to Peach)...
    * the remaining datalen will always be 36, so:
    * in[9] = 0x80; and in[10+] = 0; */
   final[0] = in[0];
   final[1] = in[1];
   final[2] = in[2];
   final[3] = in[3];
   final[4] = in[4];
   final[5] = in[5];
   final[6] = in[6];
   final[7] = in[7];
   final[8] = in[8];
   final[9] = 0x80;
   final[10] = 0;
   final[11] = 0;
   final[12] = 0;
   final[13] = 0;
   final[15] = 0;

   md5_tranform_unrolled(state, final);

   /* MD5 hash = 128 bits, zero fill remaining... */
   out[0] = ((word64 *) state)[0];
   out[1] = ((word64 *) state)[1];
   out[2] = 0;
   out[3] = 0;
}  /* end cuda_peach_md5 */

/**
 * @private
 * 160-bit Sha1 computation optimized for the Peach algorithm.
 * Places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_peach_sha1(const word32 *in, size_t inlen, word32 *out)
{
   /* SHA1 transformation constant */
   cuCONSTn860 static word32 c_k[4] =
      { 0x5a827999, 0x6ed9eba1, 0x8f1bbcdc, 0xca62c1d6 };
   /* sha1_init */
   word32 state[5] =
      { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476, 0xC3D2E1F0 };
   word32 final[16];

   final[15] = inlen << 3;
   final[15] = bswap32(final[15]);

   /* sha1_update */
   for(; inlen >= 64; inlen -= 64, in = &in[16]) {
      sha1_transform_unrolled(state, in, c_k);
   }

   /* sha1_final - somewhat conveniently (and exclusive to Peach)...
    * the remaining datalen will always be 36, so in[9] = 0x80. */
   final[0] = in[0];
   final[1] = in[1];
   final[2] = in[2];
   final[3] = in[3];
   final[4] = in[4];
   final[5] = in[5];
   final[6] = in[6];
   final[7] = in[7];
   final[8] = in[8];
   final[9] = 0x80;
   final[10] = 0;
   final[11] = 0;
   final[12] = 0;
   final[13] = 0;
   final[14] = 0;

   sha1_transform_unrolled(state, final, c_k);

   /* SHA1 hash = 160 bits, zero fill remaining... */
   out[0] = bswap32(state[0]);
   out[1] = bswap32(state[1]);
   out[2] = bswap32(state[2]);
   out[3] = bswap32(state[3]);
   out[4] = bswap32(state[4]);
   out[5] = 0;
   out[6] = 0;
   out[7] = 0;
}  /* end cu_peach_sha1() */

/**
 * @private
 * 256-bit SHA256 computation optimized for the Peach algorithm.
 * Places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_peach_sha256(const word32 *in, size_t inlen, word32 *out)
{
   /* SHA256 transformation constant */
   cuCONSTn860 static word32 c_k[64] = {
      0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
      0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
      0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
      0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
      0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
      0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
      0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
      0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
      0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
      0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
      0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
      0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
      0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
      0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
      0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
      0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
   };

   /* sha256_init */
   word32 final[16];
   word32 state[8] = {
      0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
      0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
   };

   /* prepare bitlen in final */
   final[15] = inlen << 3;
   final[15] = bswap32(final[15]);

   /* sha256_update */
   for(; inlen >= 64; inlen -= 64, in = &in[16]) {
      sha256_tranform_unrolled(state, in, c_k);
   }

   /* sha256_final - somewhat conveniently (and exclusive to Peach)...
    * the remaining datalen will always be 36, so in[9] = 0x80. */
   final[0] = in[0];
   final[1] = in[1];
   final[2] = in[2];
   final[3] = in[3];
   final[4] = in[4];
   final[5] = in[5];
   final[6] = in[6];
   final[7] = in[7];
   final[8] = in[8];
   final[9] = 0x80;
   final[10] = 0;
   final[11] = 0;
   final[12] = 0;
   final[13] = 0;
   final[14] = 0;
   sha256_tranform_unrolled(state, final, c_k);

   /* Since this implementation uses little endian byte ordering and
    * SHA uses big endian, reverse all the bytes when copying the
    * final state to the output hash. */
   out[0] = bswap32(state[0]);
   out[1] = bswap32(state[1]);
   out[2] = bswap32(state[2]);
   out[3] = bswap32(state[3]);
   out[4] = bswap32(state[4]);
   out[5] = bswap32(state[5]);
   out[6] = bswap32(state[6]);
   out[7] = bswap32(state[7]);
}  /* end cu_peach_sha256() */

/**
 * @private
 * 256-bit Sha3 (Keccak) computation optimized for the Peach algorithm.
 * Places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param keccak_final Flag indicates hash should be finalized as Keccak
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_peach_sha3(const word64 *in, size_t inlen,
   int keccak_final, word64 *out)
{
   /* Keccak permutation constant */
   cuCONSTn860 static word64 keccakf_rndc[24] = {
      WORD64_C(0x0000000000000001), WORD64_C(0x0000000000008082),
      WORD64_C(0x800000000000808a), WORD64_C(0x8000000080008000),
      WORD64_C(0x000000000000808b), WORD64_C(0x0000000080000001),
      WORD64_C(0x8000000080008081), WORD64_C(0x8000000000008009),
      WORD64_C(0x000000000000008a), WORD64_C(0x0000000000000088),
      WORD64_C(0x0000000080008009), WORD64_C(0x000000008000000a),
      WORD64_C(0x000000008000808b), WORD64_C(0x800000000000008b),
      WORD64_C(0x8000000000008089), WORD64_C(0x8000000000008003),
      WORD64_C(0x8000000000008002), WORD64_C(0x8000000000000080),
      WORD64_C(0x000000000000800a), WORD64_C(0x800000008000000a),
      WORD64_C(0x8000000080008081), WORD64_C(0x8000000000008080),
      WORD64_C(0x0000000080000001), WORD64_C(0x8000000080008008)
   };

   /* sha3_init */
   word8 state[200] = { 0 };
   word64 *st64 = (word64 *) state;
	int i;

   /* sha3_update - 136 is ctx->rsiz, fill only 17x 64-bit words in state */
   for(; inlen >= 136; inlen -= 136, in = &in[17]) {
      for (i = 0; i < 17; i++) st64[i] ^= in[i];
	   sha3_keccakf_unrolled(st64, keccakf_rndc);
   }

   /* sha3_final */
   st64[0] ^= in[0];
   st64[1] ^= in[1];
   st64[2] ^= in[2];
   st64[3] ^= in[3];
   if (inlen > PEACHGENLEN) {
      st64[4] ^= in[4];
      st64[5] ^= in[5];
      st64[6] ^= in[6];
      st64[7] ^= in[7];
      st64[8] ^= in[8];
      st64[9] ^= in[9];
      st64[10] ^= in[10];
      st64[11] ^= in[11];
      st64[12] ^= in[12];
      ((word32 *) st64)[26] ^= ((word32 *) in)[26];
      state[108] ^= keccak_final ? 0x01 : 0x06;
   } else {
      ((word32 *) st64)[8] ^= ((word32 *) in)[8];
      state[36] ^= keccak_final ? 0x01 : 0x06;
   }
   state[135] ^= 0x80;
	sha3_keccakf_unrolled(st64, keccakf_rndc);

   /* sha3_output */
   out[0] = st64[0];
   out[1] = st64[1];
   out[2] = st64[2];
   out[3] = st64[3];
}  /* end cu_peach_sha3 */

/**
 * @private
 * Perform deterministic (single precision) floating point operations on
 * @a len bytes of @a data (in 4 byte operations).
 * @param data Pointer to data to use in operations
 * @param len Length of @a data to use in operations
 * @param index Peach tile index number
 * @param txf Flag indicates @a data should be transformed by operations
 * @returns 32-bit unsigned operation code for subsequent Peach algo steps
 * @note Operations are guaranteed "deterministic" within the Peach
 * algorithm for all IEEE-754 compliant hardware on "round-to-nearest"
 * rounding mode. This is ensured by the use of CUDA intrinsics:
 * - __fdiv_rn(), __fmul_rn(), __fsub_rn(), __fadd_rn() operations, and
 * - __int2float_rn(), __uint2float_rn() conversions
*/
__device__ word32 cu_peach_dflops(void *data, size_t len,
   word32 index, int txf)
{
   cuCONSTn860 static word32 c_float[4] = {
      WORD32_C(0x26C34), WORD32_C(0x14198),
      WORD32_C(0x3D6EC), WORD32_C(0x80000000)
   };
   word8 *bp;
   float *flp, temp, flv;
   int32 operand;
   word32 op;
   unsigned i;
   word8 shift;

   /* process entire length of input data; limit to 4 byte multiples */
   /* len = len - (len & 3); // uncomment if (len % 4 != 0) is expected */
   for (op = i = 0; i < len; i += 4) {
      bp = &((word8 *) data)[i];
      if (txf) {
         /* input data is modified directly */
         flp = (float *) bp;
      } else {
         /* temp variable is modified, input data is unchanged */
         temp = *((float *) bp);
         flp = &temp;
      }
      /* first byte allocated to determine shift amount */
      shift = ((*bp & 7) + 1) << 1;
      /* remaining bytes are selected for 3 different operations based on
       * the first bytes resulting shift on precomputed contants to...
       * ... 1) determine the floating point operation type */
      op += bp[((c_float[0] >> shift) & 3)];
      /* ... 2) determine the value of the operand */
      operand = bp[((c_float[1] >> shift) & 3)];
      /* ... 3) determine the upper most bit of the operand
       *        NOTE: must be performed AFTER the allocation of the operand */
      if (bp[((c_float[2] >> shift) & 3)] & 1) operand ^= c_float[3];
      /* interpret operand as SIGNED integer and cast to float */
      flv = __int2float_rn(operand);
      /* Replace pre-operation NaN with index */
      if (isnan(*flp)) *flp = __uint2float_rn(index);
      /* Perform predetermined floating point operation */
      switch (op & 3) {
         case 3: *flp = __fdiv_rn(*flp, flv);  break;
         case 2: *flp = __fmul_rn(*flp, flv);  break;
         case 1: *flp = __fsub_rn(*flp, flv);  break;
         case 0: *flp = __fadd_rn(*flp, flv);  break;
      }
      /* Replace post-operation NaN with index */
      if (isnan(*flp)) *flp = __uint2float_rn(index);
      /* Add result of the operation to `op` as an array of bytes */
      bp = (word8 *) flp;
      op += bp[0];
      op += bp[1];
      op += bp[2];
      op += bp[3];
   }  /* end for(i = 0; ... */

   return op;
}  /* end cu_peach_dflops() */

/**
 * @private
 * Perform deterministic memory transformations on @a len bytes of @a data.
 * @param data Pointer to data to use in operations
 * @param len Length of @a data to use in operations
 * @param op Operating code from previous Peach algo steps
 * @returns 32-bit unsigned operation code for subsequent Peach algo steps
*/
__device__ word32 cu_peach_dmemtx(void *data, size_t len, word32 op)
{
   cuCONSTn860 static word64 c_flip64 = WORD64_C(0x8181818181818181);
   cuCONSTn860 static word32 c_flip32 = WORD64_C(0x81818181);
   word64 *qp = (word64 *) data;
   word32 *dp = (word32 *) data;
   word8 *bp = (word8 *) data;
   size_t len16, len32, len64, y;
   unsigned i, z;
   word8 temp;

   /* prepare memory pointers and lengths */
   len64 = (len32 = (len16 = len >> 1) >> 1) >> 1;
   /* perform memory transformations multiple times */
   for (i = 0; i < PEACHROUNDS; i++) {
      /* determine operation to use for this iteration */
      op += bp[i];
      /* select "random" transformation based on value of `op` */
      switch (op & 7) {
         case 0:  /* flip the first and last bit in every byte */
            for (z = 0; z < len64; z++) qp[z] ^= c_flip64;
            for (z <<= 1; z < len32; z++) dp[z] ^= c_flip32;
            break;
         case 1:  /* Swap bytes */
            for (y = len16, z = 0; z < len16; y++, z++) {
               temp = bp[z]; bp[z] = bp[y]; bp[y] = temp;
            }
            break;
         case 2:  /* 1's complement, all bytes */
            for (z = 0; z < len64; z++) qp[z] = ~qp[z];
            for (z <<= 1; z < len32; z++) dp[z] = ~dp[z];
            break;
         case 3:  /* Alternate +1 and -1 on all bytes */
            for (z = 0; z < len; z++) bp[z] += (z & 1) ? -1 : 1;
            break;
         case 4:  /* Alternate -i and +i on all bytes */
            for (z = 0; z < len; z++) bp[z] += (word8) ((z & 1) ? i : -i);
            break;
         case 5:  /* Replace every occurrence of 104 with 72 */ 
            for (z = 0; z < len; z++) if(bp[z] == 104) bp[z] = 72;
            break;
         case 6:  /* If byte a is > byte b, swap them. */
            for (y = len16, z = 0; z < len16; y++, z++) {
               if(bp[z] > bp[y]) {
                  temp = bp[z]; bp[z] = bp[y]; bp[y] = temp;
               }
            }
            break;
         case 7:  /* XOR all bytes */
            for (y = 0, z = 1; z < len; y++, z++) bp[z] ^= bp[y];
            break;
      } /* end switch(op & 7)... */
   } /* end for(i = 0; ... */

   return op;
}  /* end cu_peach_dmemtx() */

/**
 * @private
 * Perform Nighthash on @a inlen bytes of @a in and place result in @a out.
 * Utilizes deterministic float operations and memory transformations.
 * @param in Pointer to input data
 * @param inlen Length of data from @a in, used in non-transform steps
 * @param index Peach tile index number
 * @param txlen Length of data from @a in, used in transform steps
 * @param out Pointer to location to place resulting hash
*/
__device__ void cu_peach_nighthash(word64 *in, size_t inlen,
   word32 index, size_t txlen, word64 *out)
{
   /* Perform flops to determine initial algo type.
    * When txlen is non-zero the transformation of input data is enabled,
    * as well as the additional memory transformation process. */
   if (txlen) {
      index = cu_peach_dflops(in, txlen, index, 1);
      index = cu_peach_dmemtx(in, txlen, index);
   } else index = cu_peach_dflops(in, inlen, index, 0);

   /* reduce algorithm selection to 1 of 8 choices */
   switch (index & 7) {
      case 0: cu_peach_blake2b(in, inlen, 32, out); break;
      case 1: cu_peach_blake2b(in, inlen, 64, out); break;
      case 2: cu_peach_sha1((word32 *) in, inlen, (word32 *) out); break;
      case 3: cu_peach_sha256((word32 *) in, inlen, (word32 *) out); break;
      case 4: cu_peach_sha3(in, inlen, 0, out); break;
      case 5: cu_peach_sha3(in, inlen, 1, out); break;
      case 6: cu_peach_md2(in, inlen, out); break;
      case 7: cu_peach_md5((word32 *) in, inlen, out); break;
   }  /* end switch(algo_type)... */
}  /* end cu_peach_nighthash() */

/**
 * @private
 * Generate a tile of the Peach map.
 * @param index Index number of tile to generate
 * @param tilep Pointer to location to place generated tile
*/
__device__ void cu_peach_generate
   (word32 index, word64 *tilep, word32 *phash)
{
   int i;

   /* place initial data into seed */
   ((word32 *) tilep)[0] = index;
   ((word32 *) tilep)[1] = phash[0];
   ((word32 *) tilep)[2] = phash[1];
   ((word32 *) tilep)[3] = phash[2];
   ((word32 *) tilep)[4] = phash[3];
   ((word32 *) tilep)[5] = phash[4];
   ((word32 *) tilep)[6] = phash[5];
   ((word32 *) tilep)[7] = phash[6];
   ((word32 *) tilep)[8] = phash[7];
   /* perform initial nighthash into first row of tile */
   cu_peach_nighthash(tilep, PEACHGENLEN, index, PEACHGENLEN, tilep);
   /* fill the rest of the tile with the preceding Nighthash result */
   for (i = 0; i < (PEACHTILELEN64 - 4); i += 4) {
      tilep[i + 4] = index;
      cu_peach_nighthash(&tilep[i], PEACHGENLEN, index, SHA256LEN,
         &tilep[i + 4]);
   }
}  /* end cu_peach_generate() */

/**
 * @private
 * Perform an index jump using the hash result of the Nighthash function.
 * @param index Index number of (current) tile on Peach map
 * @param nonce Nonce for use as entropy in jump direction
 * @param tilep Pointer to tile data at @a index
 * @returns 32-bit unsigned index of next tile
*/
__device__ void cu_peach_jump(word32 *index, word64 *nonce, word64 *tilep)
{
   word64 seed[(PEACHJUMPLEN / 8) + 1];
   word32 *dp = (word32 *) seed;
   int i;

   /* construct seed for use as Nighthash input for this index on the map */
   seed[0] = nonce[0];
   seed[1] = nonce[1];
   seed[2] = nonce[2];
   seed[3] = nonce[3];
   dp[8] = *index;
#pragma unroll
   for (i = 0; i < PEACHTILELEN32; i++) {
      dp[i + 9] = ((word32 *) tilep)[i];
   }

   /* perform nighthash on PEACHJUMPLEN bytes of seed */
   cu_peach_nighthash(seed, PEACHJUMPLEN, *index, 0, seed);
   /* sum hash as 8x 32-bit unsigned integers */
   *index = dp[0] + dp[1] + dp[2] + dp[3] + dp[4] + dp[5] + dp[6] + dp[7];
   *index &= PEACHCACHELEN_M1;
}  /* end cu_peach_jump() */

/**
 * CUDA kernel for bulk generation of Peach Map tiles.
 * @param d_map Device pointer to location of Peach Map
 * @param offset Index number offset to generate tiles from
 */
__global__ void kcu_peach_build
   (word32 offset, word64 *d_map, word32 *d_phash)
{
   const word32 index = ((blockDim.x * blockIdx.x) + threadIdx.x) + offset;
   if (index < PEACHCACHELEN) {
      cu_peach_generate(index, &d_map[index * PEACHTILELEN64], d_phash);
   }
}  /* end kcu_peach_build() */

/**
 * CUDA kernel for solving a tokenized haiku as nonce output for Peach proof
 * of work. Combine haiku protocols implemented in the Trigg Algorithm with
 * the memory intensive protocols of the Peach algorithm to generate haiku
 * output as proof of work.
 * @param d_map Device pointer to Peach Map
 * @param d_ictx Device pointer to incomplete hashing contexts
 * @param d_solve Device pointer to location to place nonce on solve
*/
__global__ void kcu_peach_solve
   (word64 *d_map, SHA256_CTX *d_ictx, word8 diff, word64 *d_solve)
{
   SHA256_CTX ictx;
   word64 nonce[4];
   word8 hash[SHA256LEN];
   word32 *x, mario, tid, i;

   tid = (blockIdx.x * blockDim.x) + threadIdx.x;

   /* shift ictx to appropriate location and extract nonce */
#pragma unroll
   for (i = 0; i < sizeof(ictx) / 4; i++) {
      ((word32 *) &ictx)[i] = ((word32 *) &d_ictx[tid])[i];
   }
#pragma unroll
   for (i = 0; i < 8; i++) {
      ((word32 *) nonce)[i] = ((word32 *) &ictx.data[28])[i];
   }
   /* finalise incomplete sha256 hash */
   cu_sha256_final(&ictx, hash);
   /* initialize mario's starting index on the map, bound to PEACHCACHELEN */
   for (mario = hash[0], i = 1; i < SHA256LEN; i++) {
      mario *= hash[i];
   }
   mario &= PEACHCACHELEN_M1;
   /* perform tile jumps to find the final tile x8 */
   for (i = 0; i < PEACHROUNDS; i++) {
      cu_peach_jump(&mario, nonce, &d_map[mario * PEACHTILELEN64]);
   }
   /* hash block trailer with final tile */
   cu_sha256_init(&ictx);
   cu_sha256_update(&ictx, hash, SHA256LEN);
   cu_sha256_update(&ictx, &d_map[mario * PEACHTILELEN64], PEACHTILELEN);
   cu_sha256_final(&ictx, hash);
   /* Coarse/Fine evaluation checks */
   x = (word32 *) hash;
   for (i = diff >> 5; i; i--) if(*(x++) != 0) return;
   if (__clz(__byte_perm(*x, 0, 0x0123)) < (diff & 31)) return;

   /* check first to solve with atomic solve handling */
   if (!atomicCAS((int *) d_solve, 0, *((int *) nonce))) {
      d_solve[0] = nonce[0];
      d_solve[1] = nonce[1];
      d_solve[2] = nonce[2];
      d_solve[3] = nonce[3];
   }
}  /* end kcu_peach_solve() */

/**
 * CUDA kernel for checking Peach Proof-of-Work. The haiku must be
 * syntactically correct AND have the right vibe. Also, entropy MUST match
 * difficulty.
 * @param ictx Device pointer to incomplete hashing context
 * @param out Pointer to location to place final hash
 * @param eval Evaluation result: VEOK on success, else VERROR
*/
__global__ void kcu_peach_checkhash
   (BTRAILER *d_bt, word8 *d_out, word8 *d_eval)
{
   word64 data[(SHA256LEN + PEACHTILELEN) / 8] = { 0 };
   word64 nonce[4];
   BTRAILER *btp;
   word8 *hash = (word8 *) data;
   word64 *tile = (word64 *) &data[SHA256LEN / 8];
   word32 *x, mario;
   unsigned int tid;
   int i;

   /* init */
   tid = (blockDim.x * blockIdx.x) + threadIdx.x;
   btp = &d_bt[tid];

   /* copy nonce */
#pragma unroll
   for (i = 0; i < 8; i++) {
      ((word32 *) nonce)[i] = ((word32 *) btp->nonce)[i];
   }

   /* hash partial trailer */
   cu_sha256(btp, 124, hash);
   /* initialize mario's starting index on the map, bound to PEACHCACHELEN */
   for(mario = hash[0], i = 1; i < SHA256LEN; i++) mario *= hash[i];
   mario &= PEACHCACHELEN_M1;
   /* generate and perform tile jumps to find the final tile x8 */
   for (i = 0; i < PEACHROUNDS; i++) {
      cu_peach_generate(mario, tile, (word32 *) btp->phash);
      cu_peach_jump(&mario, nonce, tile);
   }
   /* generate the last tile */
   cu_peach_generate(mario, tile, (word32 *) btp->phash);
   /* hash bthash and final tile */
   cu_sha256(data, SHA256LEN + PEACHTILELEN, hash);
   /* pass final hash to out */
   memcpy(&d_out[SHA256LEN * tid], hash, SHA256LEN);
   /* Coarse/Fine evaluation checks */
   x = (word32 *) hash;
   i = btp->difficulty[0] >> 5;
   for (; i; i--) if(*(x++) != 0) { *d_eval = 1; return; }
   if (__clz(__byte_perm(*x, 0, 0x0123)) < (btp->difficulty[0] & 31)) {
      *d_eval = 1;
      return;
   }
}  /* end kcu_peach_checkhash() */

/**
 * Check Peach proof of work with a CUDA device.
 * Uses the first available Cuda device to check multiple POW.
 * @param count Number of block trailers to check
 * @param bt Pointer to block trailer array
 * @param out Pointer to final hash array, if non-null
 * @returns VEOK on success, else VERROR
*/
int peach_checkhash_cuda(int count, BTRAILER bt[], void *out)
{
   BTRAILER *d_bt;
   word8 *d_out, *d_eval;
   word8 eval = 0;
   int cuda_count;

   cuCHK(hipGetDeviceCount(&cuda_count), NULL, return (-1));
   if (cuda_count < 1) {
      perr("No CUDA devices...");
      return -1;
   }
   cuCHK(hipSetDevice(0), NULL, return (-1));
   cuCHK(hipMalloc(&d_bt, sizeof(BTRAILER) * count), NULL, return (-1));
   cuCHK(hipMalloc(&d_out, SHA256LEN * count), NULL, return (-1));
   cuCHK(hipMalloc(&d_eval, 1), NULL, return (-1));
   /* transfer data to device */
   cuCHK(hipMemcpy(d_bt, bt, sizeof(BTRAILER) * count,
      hipMemcpyHostToDevice), NULL, return (-1));
   cuCHK(hipMemset(d_out, 0, SHA256LEN * count), NULL, return (-1));
   cuCHK(hipMemset(d_eval, 0, 1), NULL, return (-1));
   /* launch kernel to check Peach */
   kcu_peach_checkhash<<<1, count>>>(d_bt, d_out, d_eval);
   cuCHK(hipGetLastError(), NULL, return (-1));
   /* retrieve hash/eval data */
   cuCHK(hipMemcpy(out, d_out, SHA256LEN * count,
      hipMemcpyDeviceToHost), NULL, return (-1));
   cuCHK(hipMemcpy(&eval, d_eval, 1,
      hipMemcpyDeviceToHost), NULL, return (-1));
   /* wait for device to finish */
   cuCHK(hipDeviceSynchronize(), NULL, return (-1));
   /* free memory */
   cuCHK(hipFree(d_bt), NULL, return (-1));
   cuCHK(hipFree(d_out), NULL, return (-1));
   cuCHK(hipFree(d_eval), NULL, return (-1));
   /* return */
   return (int) eval;
}  /* end peach_checkhash_cuda() */

/**
 * Free CUDA memory allocated to a previously initialized device context.
 * @param devp Pointer to DEVICE_CTX to free
 * @returns VEOK on valid DEVICE_CTX pointer, else VERROR
*/
int peach_free_cuda_device(DEVICE_CTX *devp, int status)
{
   /* check device pointer */
   if (devp == NULL) return VERROR;
   /* set device status */
   devp->status = status;
   /* free pointers -- if set */
   PEACH_CUDA_CTX *ctxp = &PeachCudaCTX[devp->id];
   if (ctxp->stream[0]) hipStreamDestroy(ctxp->stream[0]);
   if (ctxp->stream[1]) hipStreamDestroy(ctxp->stream[1]);
   if (ctxp->h_ictx[0]) hipHostFree(ctxp->h_ictx[0]);
   if (ctxp->h_ictx[1]) hipHostFree(ctxp->h_ictx[1]);
   if (ctxp->h_solve[0]) hipHostFree(ctxp->h_solve[0]);
   if (ctxp->h_solve[1]) hipHostFree(ctxp->h_solve[1]);
   if (ctxp->d_solve[0]) hipFree(ctxp->d_solve[0]);
   if (ctxp->d_solve[1]) hipFree(ctxp->d_solve[1]);
   if (ctxp->d_ictx[0]) hipFree(ctxp->d_ictx[0]);
   if (ctxp->d_ictx[1]) hipFree(ctxp->d_ictx[1]);
   if (ctxp->d_phash) hipFree(ctxp->d_phash);
   if (ctxp->d_map) hipFree(ctxp->d_map);
   /* attempt to clear last error */
   (void) hipGetLastError();

   return VEOK;
}  /* end peach_free_cuda_device() */

/**
 * (re)Initialize a device context with a CUDA device.
 * @param devp Pointer to DEVICE_CTX to initialize
 * @param id Index of CUDA device to initialize to DEVICE_CTX
 * @returns VEOK on success, else VERROR
 * @note The `id` parameter of the DEVICE_CTX must be set to an appropriate
 * CUDA device number. If not performing a re-initialization, recommend
 * using peach_init_cuda() first.
*/
int peach_init_cuda_device(DEVICE_CTX *devp, int id)
{
   static int nvml_initialized = 0;
   static unsigned nvml_count = 0;

   struct hipDeviceProp_t props;
   nvmlPciInfo_t pci;
   nvmlDevice_t *nvmlp;
   size_t ictxlen;
   unsigned i, gen, width, skip;

   if (nvml_initialized == 0) {
      /* set nvml initialized */
      nvml_initialized = 1;
      /* initialize nvml */
      if (nvmlInit() != NVML_SUCCESS) {
         nvml_count = pdebug("Unable to initialize NVML");
         plog("No NVML devices detected...");
      } else if (nvmlDeviceGetCount(&nvml_count) != NVML_SUCCESS) {
         nvml_count = pdebug("Unable to obtain NVML count");
         plog("No NVML devices detected...");
      }
   }

   devp->id = id;
   devp->work = 0;
   devp->last_work = 0;
   devp->total_work = 0;
   devp->status = DEV_NULL;
   devp->type = CUDA_DEVICE;
   nvmlp = &(PeachCudaCTX[id].nvml_device);
   /* get CUDA properties for verification with nvml device */
   if (hipGetDeviceProperties(&props, id) != hipSuccess) {
      perr("hipGetDeviceProperties(%d)", id);
   } else {
      /* scan nvml devices for match */
      PeachCudaCTX[id].nvml_enabled = 0;
      for (i = 0; i < nvml_count; i++) {
         memset(nvmlp, 0, sizeof(nvmlDevice_t));
         if (nvmlDeviceGetHandleByIndex(i, nvmlp) == NVML_SUCCESS &&
            (nvmlDeviceGetPciInfo(*nvmlp, &pci) == NVML_SUCCESS) &&
            (pci.device == props.pciDeviceID) &&
            (pci.domain == props.pciDomainID) &&
            (pci.bus == props.pciBusID)) {
            /* obtain link gen/width */
            if (nvmlDeviceGetCurrPcieLinkGeneration(*nvmlp, &gen)
                  != NVML_SUCCESS) gen = 0;
            if (nvmlDeviceGetCurrPcieLinkWidth(*nvmlp, &width)
                  != NVML_SUCCESS) width = 0;
            PeachCudaCTX[id].nvml_enabled = 1;
            break;
         }
      }
      /* store GPU name, PCI Id and gen info in nameId */
      skip = strncmp("NVIDIA ", props.name, 7) ? 0 : 7;
      snprintf(devp->nameId, sizeof(devp->nameId),
         "%04u:%02u:%02u %.128s Gen%1ux%02u", props.pciDomainID,
         props.pciDeviceID, props.pciBusID, props.name + skip, gen, width);
   }
   /* set context to CUDA id */
   cuCHK(hipSetDevice(id), devp, return VERROR);
   /* set CUDA configuration for device */
   if (hipOccupancyMaxPotentialBlockSize(&(devp->grid), &(devp->block),
         kcu_peach_solve, 0, 0) != hipSuccess) {
      pdebug("cudaOccupancy~BlockSize(%d) failed...", id);
      pdebug("Using conservative defaults for <<<512/128>>>");
      devp->grid = 512;
      devp->block = 128;
   }
   /* calculate total threads and ictxlist size */
   devp->threads = devp->grid * devp->block;
   ictxlen = sizeof(SHA256_CTX) * devp->threads;
   /* create streams for device */
   cuCHK(hipStreamCreate(&(PeachCudaCTX[id].stream[0])), devp, return VERROR);
   cuCHK(hipStreamCreate(&(PeachCudaCTX[id].stream[1])), devp, return VERROR);
   /* allocate pinned host memory for host/device transfers */
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_solve[0]), 32), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_solve[1]), 32), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_ictx[0]), ictxlen), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_ictx[1]), ictxlen), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_bt[0]), sizeof(BTRAILER)), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_bt[1]), sizeof(BTRAILER)), devp, return VERROR);
   /* allocate device memory for host/device transfers */
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_solve[0]), 32), devp, return VERROR);
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_solve[1]), 32), devp, return VERROR);
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_ictx[0]), ictxlen), devp, return VERROR);
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_ictx[1]), ictxlen), devp, return VERROR);
   /* allocate memory for Peach map on device */
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_phash), 32), devp, return VERROR);
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_map), PEACHMAPLEN), devp, return VERROR);
   /* clear device/host allocated memory */
   cuCHK(hipMemsetAsync(PeachCudaCTX[id].d_ictx[0], 0, ictxlen,
      hipStreamDefault), devp, return VERROR);
   cuCHK(hipMemsetAsync(PeachCudaCTX[id].d_ictx[1], 0, ictxlen,
      hipStreamDefault), devp, return VERROR);
   cuCHK(hipMemsetAsync(PeachCudaCTX[id].d_solve[0], 0, 32,
      hipStreamDefault), devp, return VERROR);
   cuCHK(hipMemsetAsync(PeachCudaCTX[id].d_solve[1], 0, 32,
      hipStreamDefault), devp, return VERROR);
   cuCHK(hipMemsetAsync(PeachCudaCTX[id].d_phash, 0, 32,
      hipStreamDefault), devp, return VERROR);
   memset(PeachCudaCTX[id].h_bt[0], 0, sizeof(BTRAILER));
   memset(PeachCudaCTX[id].h_bt[1], 0, sizeof(BTRAILER));
   memset(PeachCudaCTX[id].h_ictx[0], 0, ictxlen);
   memset(PeachCudaCTX[id].h_ictx[1], 0, ictxlen);
   memset(PeachCudaCTX[id].h_solve[0], 0, 32);
   memset(PeachCudaCTX[id].h_solve[1], 0, 32);

   return VEOK;
}  /* end peach_init_cuda_device() */

/**
 * Initialize a DEVICE_CTX list with CUDA devices for solving the Peach
 * proof of work algorithm.
 * @param devlist Pointer to DEVICE_CTX list to initialize
 * @param max Maximum number of CUDA devices to initialize
 * @returns number of CUDA devices available for initialization
 * @note It is possible to have "some" CUDA devices fail to initialize.
*/
int peach_init_cuda(DEVICE_CTX devlist[], int max)
{
   static int initialized = 0;
   static int num = 0;

   int id;

   /* avoid re-initialization attempts */
   if (initialized) return num;

   /* check for cuda driver and devices */
   switch (hipGetDeviceCount(&num)) {
      case hipErrorNoDevice:
         return plog("No CUDA devices detected...");
      case hipErrorInsufficientDriver:
         perr("Insufficient CUDA Driver. Update display drivers...");
         return 0;
      case hipSuccess:
         if (num > max) {
            num = max;
            plog("CUDA Devices: %d (limited)\n", num);
            perr("CUDA device count EXCEEDED maximum count parameter!");
            pwarn("Some CUDA devices will not be utilized.");
            plog("Please advise developers if this is an issue...");
         }
         break;
      default:
         perr("Unknown CUDA initialization error occured...");
         return 0;
   }

   /* set initialized */
   initialized = 1;
   if (num < 1) return (num = 0);
   /* allocate memory for PeachCudaCTX */
   PeachCudaCTX = (PEACH_CUDA_CTX *) malloc(sizeof(PEACH_CUDA_CTX) * num);
   /* initialize device contexts for CUDA num devices */
   for (id = 0; id < num; id++) peach_init_cuda_device(&devlist[id], id);

   return num;
}  /* end peach_init_cuda() */

/**
 * Try solve for a tokenized haiku as nonce output for Peach proof of work
 * on CUDA devices. Combine haiku protocols implemented in the Trigg
 * Algorithm with the memory intensive protocols of the Peach algorithm to
 * generate haiku output as proof of work.
 * @param dev Pointer to DEVICE_CTX to perform work with
 * @param bt Pointer to block trailer to solve for
 * @param diff Difficulty to test against entropy of final hash
 * @param btout Pointer to location to place solved block trailer
 * @returns VEOK on solve, VERROR on no solve, or VETIMEOUT if GPU is
 * either stopped or unrecoverable.
*/
int peach_solve_cuda(DEVICE_CTX *dev, BTRAILER *bt, word8 diff, BTRAILER *btout)
{
   int i, id, sid, grid, block, build;
   PEACH_CUDA_CTX *P;
   nvmlReturn_t nr;
   size_t ictxlen;

   /* init */
   id = dev->id;
   P = &PeachCudaCTX[id];

   /* check for GPU failure */
   if (dev->status == DEV_FAIL && dev->last_work) {
      /* recovery MAY be possible --- wait 5 seconds */
      if (difftime(time(NULL), dev->last_work) >= 5) {
         printf("CUDA#%d-> attempting failure recovery...", id);
         peach_init_cuda_device(dev, id);
      }
      return VERROR;
   }

   /* report unuseable GPUs */
   if (dev->status < DEV_NULL) return VETIMEOUT;

   /* set/check cuda device */
   cuCHK(hipSetDevice(id), dev, return VERROR);
   cuCHK(hipGetLastError(), dev, return VERROR);

   /* ensure initialization is complete */
   if (dev->status == DEV_NULL) {
      if (hipStreamQuery(hipStreamDefault) != hipSuccess) return VERROR;
      /* set next action to build Peach map */
      dev->status = DEV_INIT;
      dev->last_work = time(NULL);
      dev->total_work = 0;
      dev->work = 0;
   }

   /* build peach map */
   if (dev->status == DEV_INIT) {
      /* build peach map -- init */
      if (dev->work == 0) {
         /* ensure both streams have finished */
         if (hipStreamQuery(P->stream[1]) == hipSuccess
            && hipStreamQuery(P->stream[0]) == hipSuccess) {
            /* synchronize device before initializing new peach map */
            hipDeviceSynchronize();
            /* clear any late solves */
            cuCHK(hipMemset(P->d_solve[0], 0, 32), dev, return VERROR);
            cuCHK(hipMemset(P->d_solve[1], 0, 32), dev, return VERROR);
            memset(P->h_solve[0], 0, 32);
            memset(P->h_solve[1], 0, 32);
            /* update block trailer */
            memcpy(P->h_bt[0], bt, sizeof(BTRAILER));
            memcpy(P->h_bt[1], bt, sizeof(BTRAILER));
            /* update device phash */
            cuCHK(hipMemcpy(P->d_phash, P->h_bt[0]->phash, 32,
               hipMemcpyHostToDevice), dev, return VERROR);
            /* synchronize memory transfers before building peach map */
            hipDeviceSynchronize();
            /* flag build ready */
            build = 1;
         }
      }
      /* build peach map -- build */
      if (dev->work < PEACHCACHELEN) {
         for (sid = 0; sid < 2 && (build || dev->work > 0); sid++) {
            /* ensure stream is ready for next section of build */
            if (hipStreamQuery(P->stream[sid]) != hipSuccess) continue;
            /* set CUDA configuration for generating peach map */
            if (hipOccupancyMaxPotentialBlockSize(&grid, &block,
                  kcu_peach_build, 0, 0) != hipSuccess) {
               pdebug("cudaOccupancy~BlockSize(%d) failed...", id);
               pdebug("Using conservative defaults, <<<128/128>>>");
               grid = 128;
               block = 128;
            }
            /* launch kernel to generate map */
            kcu_peach_build<<<grid, block, 0, P->stream[sid]>>>
               ((word32) dev->work, P->d_map, P->d_phash);
            cuCHK(hipGetLastError(), dev, return VERROR);
            /* update build progress */
            dev->work += grid * block;
         }
      } else {
         /* ensure both streams have finished */
         if (hipStreamQuery(P->stream[1]) == hipSuccess
            && hipStreamQuery(P->stream[0]) == hipSuccess) {
            /* build is complete */
            dev->last_work = time(NULL);
            dev->status = DEV_IDLE;
            dev->work = 0;
         }
      }
   }

   /* check for unsolved work in block trailer */
   if (dev->status == DEV_IDLE && get32(bt->tcount)) {
      if (cmp64(bt->bnum, btout->bnum)) dev->status = DEV_WORK;
   }

   /* solve work in block trailer */
   if (dev->status == DEV_WORK) {
      for(sid = 0; sid < 2; sid++) {
         if (hipStreamQuery(P->stream[sid]) != hipSuccess) continue;
         /* check trailer for block update */
         if (memcmp(P->h_bt[sid]->phash, bt->phash, HASHLEN)) {
            dev->status = DEV_INIT;
            dev->work = 0;
            break;
         }
         /* switch to idle mode if no transactions or already solved bnum */
         if (get32(bt->tcount) == 0 || cmp64(bt->bnum, btout->bnum) == 0) {
            dev->status = DEV_IDLE;
            dev->work = 0;
            break;
         }
         /* check for solves */
         if (*(P->h_solve[sid])) {
            /* move solved nonce */
            memcpy(P->h_bt[sid]->nonce, P->h_solve[sid], 32);
            /* clear solve from host/device */
            hipMemsetAsync(P->d_solve[sid], 0, 32, P->stream[sid]);
            memset(P->h_solve[sid], 0, 32);
            /* move solved block trailer to btout */
            memcpy(btout, P->h_bt[sid], sizeof(BTRAILER));
            /* return a solve */
            return VEOK;
         }
         /* check for "on-the-fly" difficulty changes */
         diff = diff && diff < bt->difficulty[0] ? diff : bt->difficulty[0];
         /* ensure block trailer is updated */
         memcpy(P->h_bt[sid], bt, sizeof(BTRAILER));
         /* generate nonce directly into block trailer */
         trigg_generate_fast(P->h_bt[sid]->nonce);
         trigg_generate_fast(P->h_bt[sid]->nonce + 16);
         /* prepare intermediate state for next round */
         sha256_init(P->h_ictx[sid]);
         sha256_update(P->h_ictx[sid], P->h_bt[sid], 124);
         /* duplicate intermediate state with random second seed */
         for(i = 1; i < dev->threads; i++) {
            memcpy(&(P->h_ictx[sid][i]), P->h_ictx[sid], sizeof(SHA256_CTX));
            trigg_generate_fast(P->h_ictx[sid][i].data + 44);
         }
         /* transfer ictx to device */
         ictxlen = sizeof(SHA256_CTX) * dev->threads;
         hipMemcpyAsync(P->d_ictx[sid], P->h_ictx[sid], ictxlen,
            hipMemcpyHostToDevice, P->stream[sid]);
         cuCHK(hipGetLastError(), dev, return VERROR);
         /* launch kernel to solve Peach */
         kcu_peach_solve<<<dev->grid, dev->block, 0, P->stream[sid]>>>
            (P->d_map, P->d_ictx[sid], diff, P->d_solve[sid]);
         cuCHK(hipGetLastError(), dev, return VERROR);
         /* retrieve solve seed */
         hipMemcpyAsync(P->h_solve[sid], P->d_solve[sid], 32,
            hipMemcpyDeviceToHost, P->stream[sid]);
         cuCHK(hipGetLastError(), dev, return VERROR);
         /* increment progress counters */
         dev->total_work += dev->threads;
         dev->work += dev->threads;
      }
   }

   /* power and temperature monitoring (1 second interval) */
   if (P->nvml_enabled && difftime(time(NULL), dev->last_monitor)) {
      dev->last_monitor = time(NULL);
      /* get GPU device power */
      unsigned int fan;
      nr = nvmlDeviceGetFanSpeed(P->nvml_device, &fan);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) fan speed: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->fan = fan;
      /* get GPU device power */
      unsigned int power;
      nr = nvmlDeviceGetPowerUsage(P->nvml_device, &power);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) power usage: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->pow = power / 1000;
      /* get GPU device temperature */
      unsigned int temperature;
      nr = nvmlDeviceGetTemperature(P->nvml_device, NVML_TEMPERATURE_GPU,
         &temperature);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) temperature: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->temp = temperature;
      /* get GPU device utilization */
      nvmlUtilization_t utilization;
      nr = nvmlDeviceGetUtilizationRates(P->nvml_device, &utilization);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) utilization rates: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->util = utilization.gpu;
   }

   return VERROR;
}  /* end peach_solve_cuda() */

/* end include guard */
#endif
